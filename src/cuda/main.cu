#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image_write.h"

#include "types/vec3.h"
#include "types/color.h"
#include "primitives/ray.h"
#include "objects/hittable.h"
#include "objects/hittable_list.h"
#include "objects/sphere.h"
#include "objects/bvh.h"
#include "primitives/camera.h"
#include "primitives/materials/material.h"
#include "primitives/textures/texture.h"
#include "utils/rtw_stb_image.h"

#include <iostream>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <string>

// Limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= max_x) || (y >= max_y)) return;
    int pixel_index = y * max_x + x;
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(
    color *fb,
    int max_x,
    int max_y,
    int ns,
    camera **cam,
    hittable **world,
    hiprandState *rand_state
) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= max_x) || (y >= max_y)) return;
    int pixel_index = y * max_x + x;
    hiprandState local_rand_state = rand_state[pixel_index];
    color col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += (*cam)->ray_color(r, world, &local_rand_state, 10); // hardcoded max depth
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_noise_sphere(
    hittable **d_list,
    hittable **d_world,
    camera **d_camera,
    int nx,
    int ny,
    hiprandState *rand_state
) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        hiprandState local_rand_state = *rand_state;
        texture *pertext = new noise_texture(4.0f, &local_rand_state);

        d_list[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(pertext));
        d_list[1] = new sphere(vec3(0, 2, 0),  2, new lambertian(pertext));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 2);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(
            lookfrom,
            lookat,
            vec3(0, 1, 0),
            20.0,
            float(nx)/float(ny),
            aperture,
            dist_to_focus
        );
    }
}

__global__ void create_earth(
    hittable **d_list,
    hittable **d_world,
    camera **d_camera,
    const unsigned char *earth_data,
    int earth_width,
    int earth_height,
    int nx,
    int ny,
    hiprandState *rand_state
) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        hiprandState local_rand_state = *rand_state;

        texture *earth_texture = new image_texture(earth_data, earth_width, earth_height);
        d_list[0] = new sphere(vec3(0, 0, 0), 2.0f, new lambertian(earth_texture));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 1);

        vec3 lookfrom(0, 0, 12);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(
            lookfrom,
            lookat,
            vec3(0, 1, 0),
            20.0,
            float(nx)/float(ny),
            aperture,
            dist_to_focus
        );
    }
}

__global__ void create_checkered_spheres(
    hittable **d_list,
    hittable **d_world,
    camera **d_camera,
    int nx,
    int ny,
    hiprandState *rand_state
) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        hiprandState local_rand_state = *rand_state;
        texture *checker = new checker_texture(0.32f, color(.2, .3, .1), color(.9, .9, .9));
        d_list[0] = new sphere(vec3(0, -10, 0), 10.0f, new lambertian(checker));
        d_list[1] = new sphere(vec3(0, 10, 0),  10.0f, new lambertian(checker));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 2);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(
            lookfrom,
            lookat,
            vec3(0, 1, 0),
            20.0,
            float(nx)/float(ny),
            aperture,
            dist_to_focus
        );
    }
}

__global__ void create_bouncing_spheres(
    hittable **d_list,
    hittable **d_world,
    camera **d_camera,
    int nx,
    int ny,
    hiprandState *rand_state
) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        hiprandState local_rand_state = *rand_state;
        texture *checker = new checker_texture(0.32f, color(.2, .3, .1), color(.9, .9, .9));
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(checker));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                vec3 center2 = center + vec3(0, RND*0.5f, 0);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, center2, 0.2f,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, center2, 0.2f,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, center2, 0.2f, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(
            lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx)/float(ny),
            aperture,
            dist_to_focus
        );
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera, int num_objects) {
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        // Delete the 2 spheres we created
        for(int i = 0; i < num_objects; i++) {
            sphere *s = (sphere *)d_list[i];
            delete s->mat_ptr;  // Delete the material
            delete s;           // Delete the sphere
        }
        
        // Delete the hittable_list and its contents
        hittable_list *world = (hittable_list *)*d_world;
        if (world->single_object != nullptr) {
            // Delete the BVH node (this will recursively delete its children)
            delete world->single_object;
        }
        delete world;
        
        // Delete the camera
        delete *d_camera;
    }
}

int main(int argc, char *argv[]) {
    int nx = 712;
    int ny = 400;
    int ns = 50;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMallocManaged((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // We need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    int cs = 3;

    // make our world of hitables
    hittable **d_list;
    int num_objects;
    switch (cs) {
        case 1:
            num_objects = 22*22+1+3;
            break;
        case 2:
            num_objects = 2;
            break;
        case 3:
            num_objects = 2;
            break;
        case 4:
            num_objects = 1;
            break;
    }
    checkCudaErrors(hipMalloc((void **)&d_list, num_objects*sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    
    // GPU memory for earth image (only used in case 3)
    unsigned char *d_pixels = nullptr;
    
    switch (cs) {
        case 1:
            create_bouncing_spheres<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
            break;
        case 2:
            create_checkered_spheres<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
            break;
        case 3:
            create_noise_sphere<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
            break;
        case 4:
            rtw_image earth_image("earthmap.jpg");
            const unsigned char* earth_data = earth_image.flat_pixel_data();
            
            // Allocate GPU memory for the image data
            int earth_width = earth_image.width();
            int earth_height = earth_image.height();
            
            // Allocate the actual pixel data as a flat array
            checkCudaErrors(hipMalloc((void **)&d_pixels, earth_width * earth_height * 3));
            
            // Copy pixel data from CPU to GPU
            checkCudaErrors(hipMemcpy(d_pixels, earth_data, earth_width * earth_height * 3, hipMemcpyHostToDevice));
            
            create_earth<<<1, 1>>>(d_list, d_world, d_camera, d_pixels, earth_width, earth_height, nx, ny, d_rand_state2);
            
            // Note: GPU memory will be cleaned up later
            break;
    }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Time to render: " << timer_seconds << " seconds\n";

    // Output the image
    std::vector<uint8_t> image_data(nx * ny * 3);

    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            int flipped_y = ny - y - 1;
            size_t pixel_index = flipped_y * nx + x;
            int ir = int(255.999 * fb[y * nx + x].r());
            int ig = int(255.999 * fb[y * nx + x].g());
            int ib = int(255.999 * fb[y * nx + x].b());

            image_data[3 * pixel_index + 0] = ir;
            image_data[3 * pixel_index + 1] = ig;
            image_data[3 * pixel_index + 2] = ib;
        }
    }

    std::string output_path = argv[0];
    std::string executable_dir = output_path.substr(0, output_path.find_last_of('/'));
    std::string image_path = executable_dir + "\\..\\..\\..\\image_cuda.png";
    std::cout << "\nWriting image to " << image_path << std::endl;
    stbi_write_png(image_path.c_str(), nx, ny, 3, image_data.data(), nx * 3);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world, d_camera, num_objects);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));
    
    // Clean up earth image GPU memory if allocated
    if (d_pixels != nullptr) {
        checkCudaErrors(hipFree(d_pixels));
    }

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
    return 0;
}